#include <cudaMem.h>

void cudaMem::calcMem(){
  size_t mf, ma;
  hipMemGetInfo(&mf, &ma);
  memFree = mf;
  memUsed = ma-mf;
  memTotal = ma;
}

size_t cudaMem::getMemFree(){
  return memFree;
}

size_t cudaMem::getMemUsed(){
  return memUsed;
}

size_t cudaMem::getMemTotal(){
  return memTotal;
}